#include <hip/hip_runtime.h>

#include <stdlib.h>

extern
__global__ void addKernel(const int *A, const int *B, int *C, int N);


int main(int argc, char **argv) {
  int buf[3][16];
  memset(buf, 0, sizeof(buf));

  int *cuBuf;
  hipMalloc(&cuBuf, sizeof(buf));
  hipMemcpy(cuBuf, buf, sizeof(buf), hipMemcpyHostToDevice);
  dim3 grid(4), block(4);
  addKernel<<<grid,block>>>(&cuBuf[0], &cuBuf[16], &cuBuf[32], 16);


  hipFree(cuBuf);
  return 0;
}

