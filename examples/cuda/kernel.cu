#include <hip/hip_runtime.h>

__global__ void addKernel(const int *A, const int *B, int *C, int N) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if (id < N) {
    C[id] = A[id] + B[id];
  }
}

